#include "hip/hip_runtime.h"
/*
 This class represents a ballistic projectile.
*/
class projectile {
public:
	float mass; // mass, kilograms
	float area; // area, square meters (for wind resistance)
	float px,py,pz; // position, in meters
	float vx,vy,vz; // velocity, in meters/second

	__device__ void step(float dt) {
		// Start with gravity
		float ax=0.0, ay=0.0, az=-9.8;

		// Add the wind resistance
		float velocity_squared=vx*vx+vy*vy+vz*vz;
		float velocity=sqrt(velocity_squared);

		const float air_density=1.2; // kg/m^3
		const float drag_coefficient=-0.7;
		float drag=drag_coefficient*air_density*area*velocity/mass;
		ax+=drag*vx; ay+=drag*vy; az+=drag*vz;

		// Add coriolis acceleration (rotating coordinate frame)
		const float earth_rotation_speed=2*3.141592/(24*60*60); // radians/second
		float coriolis=-2*earth_rotation_speed; // around +X axis (north)
		ay+=coriolis*vz; az-=coriolis*vy;

		// Take Euler timesteps
		vx+=ax*dt; vy+=ay*dt; vz+=az*dt;
		px+=vx*dt; py+=vy*dt; pz+=vz*dt;
	}
};

__global__ void fire_one(int *arr_outta, int *arr_altitude) {
	int altitude_ceiling=100.0; // meters of altitude (see FAA regs)
    int firing = threadIdx.x + blockIdx.x*blockDim.x;

	projectile b;
	b.mass=1.5;
	b.area=0.1*0.1;
	b.px=0.0; b.py=0.0; b.pz=1.7;
	b.vx=10.0;
	b.vy=30.0;
	b.vz=firing*0.01;

	//	Move the projectile until it hits the ground, or breaches the altitude limit.
	while (true)
	{
		// Check consistency
		if (b.pz<0.0) break; // too low
		if (b.pz>altitude_ceiling) break; // too high

		// Advance physics
		b.step(0.01);
	}

	if (b.pz>altitude_ceiling)
		arr_altitude[firing]=1;
	else
		arr_altitude[firing]=0;

	if (b.py>120.0)
		arr_outta[firing]=1;
	else
		arr_outta[firing]=0;
}

long foo(void) {
	const int n_firings=10000;
	int *gpu_arr_altitude=new int[n_firings];
	int *gpu_arr_outta=new int[n_firings];

	int *cpu_arr_altitude=new int[n_firings];
	int *cpu_arr_outta=new int[n_firings];

    gpu_check(hipMalloc(&gpu_arr_altitude, n_firings*sizeof(int)));
    gpu_check(hipMalloc(&gpu_arr_outta, n_firings*sizeof(int)));

	double start=time_in_seconds();

    fire_one<<<n_firings / 256, 256>>>(gpu_arr_outta, gpu_arr_altitude);

	gpu_check(hipDeviceSynchronize()); // join
	gpu_check(hipGetLastError());

	gpu_check(hipMemcpy(cpu_arr_outta, gpu_arr_outta, n_firings*sizeof(int), hipMemcpyDeviceToHost));
	gpu_check(hipMemcpy(cpu_arr_altitude, gpu_arr_altitude, n_firings*sizeof(int), hipMemcpyDeviceToHost));

	int altitude_breaches=0, outta_park=0;

	for (int firing=0;firing<n_firings;firing++) {
		if (cpu_arr_outta[firing]) outta_park++;
		if (cpu_arr_altitude[firing]) altitude_breaches++;
	}

	double elapsed=time_in_seconds()-start;
	std::cout<<"firing: "<<elapsed*1.0e9/n_firings<<" ns/projectile\n",
	std::cout<<"outta park: "<<outta_park*100/n_firings<<"% projectiles\n";

	return altitude_breaches*100/n_firings;
}
